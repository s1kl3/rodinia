//===============================================================================================================================================================================================================200
//	INCLUDE/DEFINE
//===============================================================================================================================================================================================================200

#include "hip/hip_runtime.h"					// (in library path specified to compiler)

//===============================================================================================================================================================================================================200
//	SET DEVICE FUNCTION
//===============================================================================================================================================================================================================200

void setdevice(void)
{

	// variables
	int num_devices;
	int device;

	// work
	hipGetDeviceCount(&num_devices);
	if (num_devices > 1) {
		
		// variables
		int max_multiprocessors; 
		int max_device;
		hipDeviceProp_t properties;

		// initialize variables
		max_multiprocessors = 0;
		max_device = 0;
		
		for (device = 0; device < num_devices; device++) {
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				max_device = device;
			}
		}
		hipSetDevice(max_device);
	}

}

//===============================================================================================================================================================================================================200
//	GET LAST ERROR FUNCTION
//===============================================================================================================================================================================================================200

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		// fprintf(stderr, "Cuda error: %s: %s.\n", msg, cudaGetErrorString( err) );
		printf("Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
		fflush(NULL);
		exit(EXIT_FAILURE);
	}
}

//===============================================================================================================================================================================================================200
//	END
//===============================================================================================================================================================================================================200
